#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include <stdlib.h>
#include <assert.h> 
#include <stdio.h>
#include <time.h>

#include "edge.cuh"


#define ASSERT(exp, ...) cudaStatus = exp; \
    if (cudaStatus != hipSuccess) { \
        fprintf(stderr, __VA_ARGS__); \
        goto Error; \
    } 
