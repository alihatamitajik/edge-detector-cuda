#include "hip/hip_runtime.h"
#include "edge.cuh"

constexpr auto BLOCK_SIZE = 1024;
constexpr auto BLOCK_DIM = 32;

#define checkGpuError(exp) cudaStatus = exp; \
    if (cudaStatus != hipSuccess) { \
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(cudaStatus), __FILE__, __LINE__); \
        goto Error; \
    } 

__global__ void changeBrightnessCUDA(uint8_t* input, const int width, 
    const int height, const int brightness)
{
    int val;

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * width + j;

    if (i < height && j < width) {
        val = input[idx] + brightness;
        // Truncate the result (0..255)
        if (val > 255) {
            input[idx] = 255;
        }
        else if (val < 0) {
            input[idx] = 0;
        }
        else {
            input[idx] = val;
        }
    }
}


/* Edge detection with Sobel filter using 
 * naive image convolution operator (Implemented from psudocode of [1])
 * 
 * This function will not be used as the sobel filter we use and is just for
 * performance comparison purposes.
 * 
 * 
 * [1] https://en.wikipedia.org/wiki/Sobel_operator
 */
__global__ void sobelCUDA(const uint8_t* image, 
    const int8_t* xKernel, 
    const int8_t* yKernel,
    uint8_t* output, 
    int width, 
    int height, 
    int kernelDim, 
    int threshold)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * width + j;

    int center = (kernelDim - 1) / 2;
    float S1 = 0, S2 = 0;
    int jshift, ishift;
    int out;

    if (i >= center && j >= center && 
        i < height - center && j < width - center) 
    {
        for (int ii = 0; ii < kernelDim; ii++) {
            for (int jj = 0; jj < kernelDim; jj++) {
                jshift = jj + j - center;
                ishift = ii + i - center;
                S1 += image[ishift * width + jshift] * xKernel[ii * kernelDim + jj];
                S2 += image[ishift * width + jshift] * yKernel[ii * kernelDim + jj];
            }
        }

        out = sqrtf(S1 * S1 + S2 * S2);
        output[idx] = out > threshold ? out : 0;
    }
}

hipError_t naiveSobel(uint8_t* dev_input, uint8_t* dev_edge, 
    int width, int height, int threshold) 
{
    hipError_t cudaStatus;

    // Kernels are aligned properly so we only should multiply values
    int8_t xKernel[][3] = { 
        {-1, 0, 1},
        {-2, 0, 2}, 
        {-1, 0, 1} 
    };

    int8_t yKernel[][3] = { 
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1} 
    };

    dim3 block(BLOCK_DIM, BLOCK_DIM);
    dim3 grid(width / BLOCK_DIM + 1, height / BLOCK_DIM + 1);

    int8_t* dev_xK;
    int8_t* dev_yK;
    size_t kernelSize = 3 * 3 * sizeof(int8_t);

    checkGpuError(hipMalloc((void**)&dev_xK, kernelSize));

    checkGpuError(hipMalloc((void**)&dev_yK, kernelSize));

    checkGpuError(hipMemcpy(dev_xK, xKernel, kernelSize, hipMemcpyHostToDevice));

    checkGpuError(hipMemcpy(dev_yK, yKernel, kernelSize, hipMemcpyHostToDevice));

    sobelCUDA<<<grid, block>>>(dev_input, dev_xK, dev_yK, dev_edge,
        width, height, 3, threshold);

    checkGpuError(hipGetLastError());

    // Wait untill Data is ready
    checkGpuError(hipDeviceSynchronize());

Error:
    hipFree(xKernel);
    hipFree(yKernel);
    return cudaStatus;
}

/*
 * Hard-Coded Sobel Filter in the kernel.
 * 
 * In this implementation, we use the properties of the algorithm (make use of 
 * zeros in the filters, Loop unrowling, using subtraction instead of "*-1" and
 * reducing kernel launches.
 * 
 * The above strategies boosts the performance of the code!
 * 
 */

__global__ void sobelOptimizedCUDA(const uint8_t* image, uint8_t* output,
    int width, int height, int threshold)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int idx = i * width + j;

    float S1, S2;
    int out;
    if (j > 0 && i > 0 && j < width - 1 && i < height - 1) {
        S1 = image[(i - 1) * width + (j + 1)] - image[(i - 1) * width + (j - 1)] - image[(i + 1) * width + (j - 1)] +
             2 * (image[i * width + (j + 1)] - image[i * width + (j - 1)]) + (image[(i + 1) * width + (j + 1)]);

        S2 = (image[(i - 1) * width + (j - 1)]) + (2 * image[(i - 1) * width + j]) + (image[(i - 1) * width + (j + 1)]) +
            (-1 * image[(i + 1) * width + (j - 1)]) + (-2 * image[(i + 1) * width + j]) + (-1 * image[(i + 1) * width + (j + 1)]);


        out = sqrtf(S1 * S1 + S2 * S2);
        output[idx] = out > threshold ? out : 0;
    }
}

/*
 * Hard-Coded + Shared Memory
 * 
 * Also we will access the memory way less with shared data and also helps the 
 * bandwidth. 
 */
__global__ void sobelOptimizedShCUDA(const uint8_t* image, uint8_t* output,
    int width, int height, int threshold)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * width + j;
    int out;
    float S1, S2;

    __shared__ uint8_t sdata[34][34];
    if (i < height && j < width) {
        // Main data
        sdata[threadIdx.y + 1][threadIdx.x + 1] = image[idx];
        
        // Boudnries
        if (threadIdx.y == 0 && blockIdx.y != 0) {
            sdata[0][threadIdx.x + 1] = image[(i - 1) * width + j];
            if (threadIdx.x == 0 && blockIdx.x != 0) {
                sdata[0][0] = image[(i - 1) * width + (j-1)];
            }
        }
        if (threadIdx.x == 0 && blockIdx.x != 0) {
            sdata[threadIdx.y + 1][0] = image[(i) * width + j - 1];
            if (threadIdx.y == 31 && i != height - 1) {
                sdata[33][0] = image[(i + 1) * width + j - 1];
            }
        }
        if (threadIdx.y == 31 && i != height - 1) {
            sdata[33][threadIdx.x + 1] = image[((i + 1) * width + j)];
            if (threadIdx.x == 31 && j != width - 1) {
                sdata[33][33] = image[(i + 1) * width + j + 1];
            }
        }
        if (threadIdx.x == 31 && j != width - 1) {
            sdata[threadIdx.y + 1][33] = image[i * width + j + 1];
            if (threadIdx.y == 0 && blockIdx.y != 0) {
                sdata[0][33] = image[(i - 1) * width + j + 1];
            }
        }
    }
    // waits untill shared data is completed
    __syncthreads();

    if (idx == 0) {
        for (int ii = 0; ii < 33; ii++) {
            for (int jj = 0; jj < 33; jj++) {
                if (sdata[ii + 1][jj + 1] != image[ii * width + jj]) {
                    printf("%d, %d::%d, %d\n", blockIdx.x, blockIdx.y, ii, jj);
                }
            }
        }
    }

    if (i >= 1 && j >= 1 &&
        i < height - 1 && j < width - 1)
    { 
        S1 = sdata[threadIdx.y][threadIdx.x + 2] - sdata[threadIdx.y][threadIdx.x]
            + 2 * (sdata[threadIdx.y + 1][threadIdx.x + 2] - sdata[threadIdx.y + 1][threadIdx.x])
            + sdata[threadIdx.y + 2][threadIdx.x + 2] - sdata[threadIdx.y + 2][threadIdx.x];

        S2 = sdata[threadIdx.y + 2][threadIdx.x + 2] + sdata[threadIdx.y + 2][threadIdx.x] 
            + 2 * (sdata[threadIdx.y + 2][threadIdx.x + 1] - sdata[threadIdx.y][threadIdx.x + 1])
            - sdata[threadIdx.y][threadIdx.x + 2] - sdata[threadIdx.y][threadIdx.x];


        out = sqrtf(S1 * S1 + S2 * S2);
        output[idx] = out > threshold ? out : 0;
    }
}

__host__ hipError_t launchDetectEdge(uint8_t * input, uint8_t * bright, uint8_t * edge,
    int width, int height, int brightness, int threshold)
{
    hipError_t cudaStatus;
    uint8_t* dev_input;
    uint8_t* dev_edge;
    size_t imageSize = width * height * sizeof(uint8_t);
    dim3 block(BLOCK_DIM, BLOCK_DIM);
    dim3 grid(width/BLOCK_DIM + (width%BLOCK_DIM!=0), 
        height/BLOCK_DIM + (width % BLOCK_DIM != 0));

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkGpuError(hipSetDevice(0));

    checkGpuError(hipMalloc((void**)&dev_input, imageSize));

    checkGpuError(hipMalloc((void**)&dev_edge, imageSize));

    checkGpuError(hipMemcpy(dev_input, input, imageSize, hipMemcpyHostToDevice));


    changeBrightnessCUDA <<<grid, block>>> (dev_input, width, height, brightness);
    checkGpuError(hipGetLastError());
    checkGpuError(hipDeviceSynchronize());


    checkGpuError(hipMemcpyAsync(bright, dev_input, imageSize, hipMemcpyDeviceToHost));


    //checkGpuError(naiveSobel(dev_input, dev_edge, width, height, threshold));

    sobelOptimizedShCUDA <<<grid, block>>> (dev_input, dev_edge, width, height, threshold);
    checkGpuError(hipGetLastError());
    checkGpuError(hipDeviceSynchronize());

    checkGpuError(hipMemcpy(edge, dev_edge, imageSize, hipMemcpyDeviceToHost));

Error:
    hipFree(dev_input);
    hipFree(dev_edge);
    return cudaStatus;
}